#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <algorithm>

#include "cuda/warp/cu_random.cuh"
#include "../numerical_benchmark/cuda/cuda_benchmarks_warp.cuh"
#include "cuda/warp/strategies.cuh"
#include "../common/cuvector.cuh"
#include "../common/cuda_err.cuh"
#include "../common/helper_functions.hpp"
#include "cuda/solution.cuh"
#include "cuda/warp/selection.cuh"

namespace cg = cooperative_groups;

__host__ __device__ __forceinline__ int log2_ceil(int value)
{
    int log2_value = 0;
    while ((1 << log2_value) < value)
        ++log2_value;
    return log2_value;
}

__host__ __device__ __forceinline__ int next_power_of_two(int x)
{
    int log2_elements = log2_ceil(x);
    return 1 << log2_elements;
}

template <int tile_sz>
__device__ __forceinline__ void initialise(
    const cg::thread_block_tile<tile_sz> &g,
    cu_random<tile_sz> &rnd,
    float *__restrict__ x,
    const int dim,
    const float lower_bound,
    const float upper_bound)
{
    for (int i = g.thread_rank(); i < dim; i += g.size())
        x[i] = rnd.uniform(lower_bound, upper_bound);
    g.sync();
}

template <int tile_sz>
__device__ __forceinline__ void replacement(
    const cg::thread_block_tile<tile_sz> &g,
    float *__restrict__ x,
    float *__restrict__ fitness,
    float *__restrict__ offspring,
    float *__restrict__ offspring_fitness,
    const int dim,
    solution *sol)
{
    if (*offspring_fitness < *fitness)
    {
        for (int i = g.thread_rank(); i < dim; i += g.size())
            x[i] = offspring[i];
        if (g.thread_rank() == 0)
        {
            *fitness = *offspring_fitness;
            check_solution(fitness, sol, g.meta_group_rank());
        }
    }
    g.sync();
}

/***************************
 * DIFFERENTIAL EVOLUTION  *
 ****************************/
template <int tile_sz>
__global__ void differential_evolution_kernel(
    int function,
    float *__restrict__ rotate,
    float *__restrict__ shift,
    int *__restrict__ shuffle,
    float *__restrict__ population,
    float *__restrict__ fitness,
    float *__restrict__ migrants,
    float *__restrict__ migrant_fitness,
    const float F,
    const float CR,
    const int num_migrants,
    const int dim,
    const int popsize,
    const float lower_bound,
    const float upper_bound,
    const int evaluations,
    const int migration,
    hiprandState *__restrict__ states,
    solution *sol)
{
    const auto thread_block = cg::this_thread_block();
    const auto g = cg::tiled_partition<tile_sz>(thread_block); // splits thread_block into thread block tiles of tile_sz
    const int block_tid = thread_block.thread_rank();          // overall thread number
    const int island_idx = g.meta_group_rank();                // population individual number
    if (island_idx >= popsize)                                 // any extra threads return
        return;

    // population and fitness values in global memory
    float *island_population = &population[blockIdx.x * popsize * dim];
    float *island_fitness = &fitness[blockIdx.x * popsize];

    // setup shared memory
    extern __shared__ char shmem[];
    int memory_size = next_power_of_two(dim);
    float *y = (float *)shmem;                      // dynamically allocated shared memory
    float *this_y = &y[island_idx * memory_size];   // used for objective functions
    float *this_z = &this_y[popsize * memory_size]; // used for objective functions
    float *this_offspring = &this_z[popsize * memory_size];
    float *this_offspring_fitness = &y[3 * popsize * memory_size + island_idx];

    cu_random<tile_sz> rnd(&states[blockIdx.x * blockDim.x + block_tid]); // warp based random class

    if (migration == 0) // initialise population
    {
        initialise<tile_sz>(g, rnd, &island_population[island_idx * dim], dim, lower_bound, upper_bound);
        benchmarks_warp::evaluate<tile_sz>(g, function, &island_population[island_idx * dim], this_y, this_z, &island_fitness[island_idx], dim, shift, rotate, shuffle);
        if (g.thread_rank() == 0)
            check_solution(&island_fitness[island_idx], sol, island_idx);
    }

    thread_block.sync();

    for (int i = 1; i <= evaluations && !sol->solution_found; i++) // evaluation loop
    {
        strategies_warp::rand_one_binary<tile_sz>(g, rnd, island_population, this_offspring, dim, popsize, CR, F, lower_bound, upper_bound);
        benchmarks_warp::evaluate<tile_sz>(g, function, this_offspring, this_y, this_z, this_offspring_fitness, dim, shift, rotate, shuffle);
        thread_block.sync();
        replacement<tile_sz>(g, &island_population[island_idx * dim], &island_fitness[island_idx], this_offspring, this_offspring_fitness, dim, sol);
        thread_block.sync();
    }

    if (island_idx < num_migrants) // update migrants
    {
        int migrant_idx = blockIdx.x * num_migrants + island_idx;
        int good_idx = selection_warp::tournament_selection<tile_sz>(g, rnd, island_population, island_fitness, dim, popsize, 2, num_migrants, false);
        g.sync();
        for (int i = g.thread_rank(); i < dim; i += g.size())
            migrants[migrant_idx * dim + i] = island_population[good_idx * dim + i];
        if (g.thread_rank() == 0)
            migrant_fitness[migrant_idx] = island_fitness[good_idx];
    }
}

template <int tile_sz>
__global__ void migration_kernel(
    float *__restrict__ population,
    float *__restrict__ fitness,
    float *__restrict__ migrants,
    float *__restrict__ migrant_fitness,
    const int num_migrants,
    const int dim,
    const int popsize,
    const int islands,
    hiprandState *states)
{
    const auto thread_block = cg::this_thread_block();
    const auto g = cg::tiled_partition<tile_sz>(thread_block); // splits thread_block into thread block tiles of tile_sz
    const int block_tid = thread_block.thread_rank(); // overall thread number
    const int island_idx = g.meta_group_rank(); // population individual number
    if (island_idx >= popsize) // any extra threads return
        return;
    cu_random<tile_sz> rnd(&states[blockIdx.x * blockDim.x + block_tid]); // warp based random class

    int next_island = (blockIdx.x + 1) % islands; // next island in unidirectional ring 

    float *island_population = &population[blockIdx.x * popsize * dim];
    float *island_fitness = &fitness[blockIdx.x * popsize];

    if (island_idx < num_migrants)
    {
        int migrant_idx = next_island * num_migrants + island_idx;
        int replace_idx = selection_warp::tournament_selection<tile_sz>(g, rnd, island_population, island_fitness, dim, popsize, 2, num_migrants, true);
        g.sync();
        for (int i = g.thread_rank(); i < dim; i += g.size())
            island_population[replace_idx * dim + i] = migrants[migrant_idx * dim + i];
        if (g.thread_rank() == 0)
            island_fitness[replace_idx] = migrant_fitness[migrant_idx];
    }
}

int main(int argc, char **argv)
{
    if (argc != 8)
    {
        std::cout << "Usage: #OBJFUNC #DIMENSION #POPSIZE #ISLANDS #RUNS #EVAL_BET_MIG PLOT" << std::endl;
        exit(0);
    }
    int function_number = atoi(argv[1]);
    const int dim = atoi(argv[2]);
    int island_popsize = atoi(argv[3]);
    int threads = 32;
    // if (island_popsize == 32)
    //     threads = 16;
    // else
    //     threads = 32;
    int islands = atoi(argv[4]);
    int runs = atoi(argv[5]);
    int eval_between_migration = atoi(argv[6]);
    int plot_data = atoi(argv[7]);
    int evaluations;
    if (dim == 50)
        evaluations = 5000000;
    else if (dim == 100)
        evaluations = 10000000;
    const float lower_bound = -100.0f;
    const float upper_bound = 100.0f;

    int num_migrants = island_popsize / 10 + 1;
    const float F = 0.5;
    const float CR = 0.3;

    // set the CUDA device, in this case the first GPU
    int device = 0;
    hipDeviceProp_t device_prop;
    cuda_error_check(hipGetDeviceProperties(&device_prop, device));
    cuda_error_check(hipSetDevice(device));

    cuBenchmarkData cuBD(function_number, dim);

    const unsigned long seed = 0;
    
    // create launch parameters for CUDA kernels
    dim3 grid_size(islands);
    dim3 block_size(threads * island_popsize);
    dim3 migrant_block_size(threads * num_migrants);

    // initialise CPU side memory
    std::vector<float> best_log;
    std::vector<float> time_log;
    std::vector<int> eval_log;

    // initialise data structures for CPU and CUDA data
    cuvector<float> population(island_popsize * islands * dim);
    cuvector<float> fitness(island_popsize * islands);

    solution h_solution, *d_solution;
    cuda_error_check(hipMalloc((void **)&d_solution, sizeof(solution)));

    // initialise memory for CUDA data
    hiprandState *d_random_states;
    cuda_error_check(hipMalloc((void **)&d_random_states, island_popsize * islands * threads * sizeof(hiprandState)));
    float *d_migrants, *d_migrant_fitness;
    cuda_error_check(hipMalloc((void **)&d_migrants, num_migrants * islands * dim * sizeof(float)));
    cuda_error_check(hipMalloc((void **)&d_migrant_fitness, num_migrants * islands * sizeof(float)));

    // initialise the device random states
    initialise_curand_kernel<<<islands, block_size>>>(d_random_states, seed);
    cuda_error_check(hipPeekAtLastError());
    cuda_error_check(hipDeviceSynchronize());

    for (int i = 0; i < runs; i++)
    {
        // reset solution
        cuda_error_check(hipMemset(d_solution, 0, sizeof(solution)));

        std::ofstream("data/all_pop_data.txt"); // this will wipe the files
        std::ofstream("data/all_fitness_data.txt");
        // now open them in append mode
        auto population_file = std::ofstream("data/all_pop_data.txt", std::ofstream::out | std::ofstream::app); // now open in append mode
        auto fitness_file = std::ofstream("data/all_fitness_data.txt", std::ofstream::out | std::ofstream::app);

        std::vector<std::vector<float>> fitness_vector(islands);

        int evals = 0;
        int m = 0;
        size_t shared_bytes = (island_popsize + island_popsize * next_power_of_two(dim) * 3) * sizeof(float);

        // if we need more more shared memory than default, request more
        // std::cout << "Require: " << shared_bytes << " and default is " << device_prop.sharedMemPerBlock << std::endl; 
        // std::cout << device_prop.sharedMemPerBlockOptin << std::endl;
        if (shared_bytes >= device_prop.sharedMemPerBlock)
        {
            switch (threads)
            {
            case 32:
                cuda_error_check(hipFuncSetAttribute(reinterpret_cast<const void*>(differential_evolution_kernel<32>), hipFuncAttributeMaxDynamicSharedMemorySize, shared_bytes));
                break;
            case 16:
                cuda_error_check(hipFuncSetAttribute(reinterpret_cast<const void*>(differential_evolution_kernel<16>), hipFuncAttributeMaxDynamicSharedMemorySize, shared_bytes));
                break;
            case 8:
                cuda_error_check(hipFuncSetAttribute(reinterpret_cast<const void*>(differential_evolution_kernel<8>), hipFuncAttributeMaxDynamicSharedMemorySize, shared_bytes));
                break;
            }
        }

        // setup cuda timer to time the algorithm
        float time_ms;
        hipEvent_t start, stop;
        cuda_error_check(hipEventCreate(&start));
        cuda_error_check(hipEventCreate(&stop));
        cuda_error_check(hipEventRecord(start, 0));

        do
        {
            switch (threads)
            {
            case 32:
                differential_evolution_kernel<32><<<grid_size, block_size, shared_bytes>>>(
                    function_number,
                    cuBD.get_rotate_transpose_ptr(),
                    cuBD.get_shift_ptr(),
                    cuBD.get_shuffle_ptr(),
                    population.get_device_ptr(),
                    fitness.get_device_ptr(),
                    d_migrants,
                    d_migrant_fitness,
                    F,
                    CR,
                    num_migrants,
                    dim,
                    island_popsize,
                    lower_bound,
                    upper_bound,
                    eval_between_migration,
                    m,
                    d_random_states,
                    d_solution);
                break;
            case 16:
                differential_evolution_kernel<16><<<grid_size, block_size, shared_bytes>>>(
                    function_number,
                    cuBD.get_rotate_transpose_ptr(),
                    cuBD.get_shift_ptr(),
                    cuBD.get_shuffle_ptr(),
                    population.get_device_ptr(),
                    fitness.get_device_ptr(),
                    d_migrants,
                    d_migrant_fitness,
                    F,
                    CR,
                    num_migrants,
                    dim,
                    island_popsize,
                    lower_bound,
                    upper_bound,
                    eval_between_migration,
                    m,
                    d_random_states,
                    d_solution);
                break;
            case 8:
                differential_evolution_kernel<8><<<grid_size, block_size, shared_bytes>>>(
                    function_number,
                    cuBD.get_rotate_transpose_ptr(),
                    cuBD.get_shift_ptr(),
                    cuBD.get_shuffle_ptr(),
                    population.get_device_ptr(),
                    fitness.get_device_ptr(),
                    d_migrants,
                    d_migrant_fitness,
                    F,
                    CR,
                    num_migrants,
                    dim,
                    island_popsize,
                    lower_bound,
                    upper_bound,
                    eval_between_migration,
                    m,
                    d_random_states,
                    d_solution);
                break;
            }

            cuda_error_check(hipPeekAtLastError());
            cuda_error_check(hipDeviceSynchronize());

            if (plot_data)
            {
                // copy population and statistical data from gpu to cpu
                fitness.cpu();
                population.cpu();

                for (int j = 0; j < islands; j++)
                {
                    fitness_file << function_number << " " << j << " " << island_popsize << " ";
                    for (int k = 0; k < island_popsize; k++)
                        fitness_file << fitness[j * island_popsize + k] << " ";
                    fitness_file << std::endl;

                    population_file << function_number << " " << j << " " << island_popsize << " " << dim << " ";
                    for (int k = 0; k < island_popsize; k++)
                        for (int p = 0; p < dim; p++)
                            population_file << population[j * island_popsize * dim + k * dim + p] << " ";
                    population_file << std::endl;
                }
            }
            // grab solution struct and see whether we stop
            cuda_error_check(hipMemcpy(&h_solution, d_solution, sizeof(solution), hipMemcpyDeviceToHost));
            if (h_solution.solution_found > 0)
            {
                evals += h_solution.solution_iteration * island_popsize * islands;
                break;
            }
            evals += eval_between_migration * island_popsize * islands;
            m++;
            if (islands > 1 && evals < evaluations)
            {
                switch (threads)
                {
                case 32:
                    migration_kernel<32><<<grid_size, block_size>>>(
                        population.get_device_ptr(),
                        fitness.get_device_ptr(),
                        d_migrants,
                        d_migrant_fitness,
                        num_migrants,
                        dim,
                        island_popsize,
                        islands,
                        d_random_states);
                    break;
                case 16:
                    migration_kernel<16><<<grid_size, block_size>>>(
                        population.get_device_ptr(),
                        fitness.get_device_ptr(),
                        d_migrants,
                        d_migrant_fitness,
                        num_migrants,
                        dim,
                        island_popsize,
                        islands,
                        d_random_states);
                    break;
                case 8:
                    migration_kernel<8><<<grid_size, block_size>>>(
                        population.get_device_ptr(),
                        fitness.get_device_ptr(),
                        d_migrants,
                        d_migrant_fitness,
                        num_migrants,
                        dim,
                        island_popsize,
                        islands,
                        d_random_states);
                    break;
                }
                cuda_error_check(hipPeekAtLastError());
                cuda_error_check(hipDeviceSynchronize());
            }
        } while (evals < evaluations);

        // stop the timer
        cuda_error_check(hipEventRecord(stop, 0));
        cuda_error_check(hipEventSynchronize(stop));
        cuda_error_check(hipEventElapsedTime(&time_ms, start, stop));
        auto time_sec = time_ms / 1000;

        // get population data
        population.cpu();
        fitness.cpu();

        auto best = *std::min_element(fitness.begin(), fitness.end());
        std::cout << "Iteration: " << i << " Function: " << function_number << " Best fitness: " << best << " Time: " << time_sec << std::endl;
        best_log.push_back(best);
        time_log.push_back(time_sec);
        eval_log.push_back(evals);
    }
    auto min_fitness = *std::min_element(best_log.begin(), best_log.end());
    auto max_fitness = *std::max_element(best_log.begin(), best_log.end());

    log_data(function_number, dim, island_popsize, islands, "degi_16", best_log, time_log, eval_log);

    float mean, stdev;
    mean_and_stdev(best_log, mean, stdev);
    float time_mean, time_stdev;
    mean_and_stdev(time_log, time_mean, time_stdev);
    float gen_mean, gen_stdev;
    mean_and_stdev(eval_log, gen_mean, gen_stdev);

    std::cout << function_number << ", " << min_fitness << ", " << max_fitness << ", " << mean
              << ", " << stdev << ", " << time_mean << ", " << time_stdev << ", "
              << gen_mean << ", " << gen_stdev << std::endl;
    hipFree(d_random_states);
    hipFree(d_migrants);
    hipFree(d_migrant_fitness);
}