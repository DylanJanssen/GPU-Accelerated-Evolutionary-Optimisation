#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cudaProfiler.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <numeric>
#include <chrono>

#include "../common/helper_functions.hpp"
#include "../common/cuda_err.cuh"
#include "../common/cuvector.cuh"
#include "../numerical_benchmark/cuda/cuda_benchmarks_block.cuh"
#include "cuda/solution.cuh"
#include "cuda/block/random.cuh"

namespace cg = cooperative_groups;

__device__ __forceinline__ void population_initialisation(
    const cg::thread_block &g,
    float *__restrict__ population,
    const int dim,
    const int popsize,
    const float lower_bound,
    const float upper_bound,
    hiprandState *__restrict__ states)
{
    if (g.group_index().x >= popsize)
        return;
    float *this_x = population + blockIdx.x * dim;
    hiprandState *state = &states[blockIdx.x * blockDim.x + threadIdx.x];

    for (int i = threadIdx.x; i < dim; i += blockDim.x)
        this_x[i] = lower_bound + hiprand_uniform(state) * (upper_bound - lower_bound);
    g.sync();
}

__global__ void select_random_trial_vectors(
    int *__restrict__ indices,
    const int popsize,
    hiprandState *__restrict__ states)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= popsize)
        return;
    hiprandState *state = &states[blockIdx.x * blockDim.x + threadIdx.x];
    int a = mutually_exclusive_random_int(0, popsize, state);
    int b = mutually_exclusive_random_int(0, popsize, state, a);
    int c = mutually_exclusive_random_int(0, popsize, state, a, b);
    indices[idx] = a;
    indices[idx + popsize] = b;
    indices[idx + popsize + popsize] = c;
}

__device__ __forceinline__ void trial_vector_generation(
    const cg::thread_block &g,
    float *__restrict__ population,
    float *__restrict__ this_offspring,
    const int dim,
    const int popsize,
    int *__restrict__ indices,
    const float crossover_rate,
    const float f,
    const float lower_bound,
    const float upper_bound,
    hiprandState *__restrict__ states)
{
    int idx = g.group_index().x;
    if (idx >= popsize)
        return;
    // float *this_offspring = &offspring[idx * dim];
    int a = indices[idx];
    int b = indices[idx + popsize];
    int c = indices[idx + popsize + popsize];
    hiprandState *state = &states[blockIdx.x * blockDim.x + threadIdx.x];
    __shared__ int j;
    if (g.thread_rank() == 0)
        j = mutually_exclusive_random_int(0, dim, state);
    g.sync();
    for (int i = g.thread_rank(); i < dim; i += g.size())
        if (hiprand_uniform(state) < crossover_rate || i == j)
        {
            this_offspring[i] = population[a * dim + i] + f * (population[b * dim + i] - population[c * dim + i]);
            if (this_offspring[i] < lower_bound || this_offspring[i] > upper_bound)
                this_offspring[i] = lower_bound + hiprand_uniform(state) * (upper_bound - lower_bound);
        }
        else
            this_offspring[i] = population[idx * dim + i];
    g.sync();
}

__device__ __forceinline__ void replacement(
    const cg::thread_block &g,
    float *__restrict__ this_x,
    float *__restrict__ this_fitness,
    float *__restrict__ this_offspring,
    float *__restrict__ this_offspring_fitness,
    const int dim,
    const int popsize,
    solution *sol)
{
    int idx = g.group_index().x;
    if (idx >= popsize)
        return;
    if (*this_offspring_fitness < *this_fitness)
    {
        for (int i = g.thread_rank(); i < dim; i += g.size())
            this_x[i] = this_offspring[i];
        if (g.thread_rank() == 0)
        {
            *this_fitness = *this_offspring_fitness;
            check_solution(this_fitness, sol, idx);
        }
    }
    g.sync();
}

__global__ void population_initialisation_and_function_evaluation(
    int function_number,
    float *__restrict__ x,
    float *__restrict__ fitness,
    const int dim,
    const int popsize,
    const float lower_bound,
    const float upper_bound,
    float *__restrict__ shift_data,
    float *__restrict__ rotate_data,
    int *__restrict__ shuffle_data,
    hiprandState *__restrict__ states,
    solution *sol)
{
    extern __shared__ char shmem[];
    float *y = (float *)shmem;
    float *z = &y[next_power_of_two(dim)];
    cg::thread_block g = cg::this_thread_block();
    float *this_x = x + blockIdx.x * dim;
    float *this_fitness = fitness + blockIdx.x;

    population_initialisation(g, x, dim, popsize, lower_bound, upper_bound, states);
    function_evaluation(g, function_number, this_x, y, z, this_fitness, dim, shift_data, rotate_data, shuffle_data, true, true);
    if (threadIdx.x == 0)
        check_solution(this_fitness, sol, blockIdx.x);
}

__global__ void trial_vector_generation_function_evaluation_and_replacement(
    int function_number,
    float *__restrict__ x,
    float *__restrict__ fitness,
    float *__restrict__ offspring,
    float *__restrict__ offspring_fitness,
    const int dim,
    const int popsize,
    int *__restrict__ indices,
    const float crossover_rate,
    const float f,
    const float lower_bound,
    const float upper_bound,
    float *__restrict__ shift_data,
    float *__restrict__ rotate_data,
    int *__restrict__ shuffle_data,
    hiprandState *__restrict__ states,
    solution *sol)
{
    extern __shared__ char shmem[];
    float *y = (float *)shmem;
    float *z = &y[next_power_of_two(dim)];
    float *this_offspring = &z[next_power_of_two(dim)];
    float *this_offspring_fitness = &this_offspring[next_power_of_two(dim)];
    cg::thread_block g = cg::this_thread_block();
    float *this_x = x + blockIdx.x * dim;
    float *this_fitness = fitness + blockIdx.x;
    trial_vector_generation(g, x, this_offspring, dim, popsize, indices, crossover_rate, f, lower_bound, upper_bound, states);
    function_evaluation(g, function_number, this_offspring, y, z, this_offspring_fitness, dim, shift_data, rotate_data, shuffle_data, true, true);
    replacement(g, this_x, this_fitness, this_offspring, this_offspring_fitness, dim, popsize, sol);
}

int main(int argc, char **argv)
{
    if (argc != 5)
    {
        std::cout << "Usage: ./program_name #OBJFUNC #DIMENSION #POPSIZE #RUNS" << std::endl;
        exit(0);
    }
    const int function_number = stoi(argv[1]);
    const int dim = stoi(argv[2]);
    const int popsize = stoi(argv[3]);
    const int total_runs = stoi(argv[4]);

    int max_evaluations = 10000000; 
    if (dim == 50)
        max_evaluations = 5000000;

    std::cout << "Running all-gpu ide with popsize=" << popsize << std::endl; 

    const float lower_bound = -100.0f;
    const float upper_bound = 100.0f;
    const float F = 0.5;
    const float CR = 0.3;

    cuvector<float> x(popsize * dim);
    cuvector<float> fitness(popsize);
    cuvector<float> offspring(popsize * dim);
    cuvector<float> offspring_fitness(popsize);
    hiprandState *d_random_states[2];
    int threads = next_power_of_two(dim);
    int blocks = popsize;
    cuda_error_check(hipMalloc((void **)&d_random_states[0], blocks * threads * sizeof(hiprandState)));
    const unsigned long seed = 0;
    initialise_curand_kernel<<<blocks, threads>>>(d_random_states[0], seed);
    cuda_error_check(hipPeekAtLastError());
    cuda_error_check(hipDeviceSynchronize());

    hipStream_t stream[3];
    for (int i = 0; i < 3; i++)
        hipStreamCreate(&stream[i]);

    // memory for random indices
    int *indices[2];
    int req_indices = 3;
    int n_threads = 32;
    int n_blocks = int_divide_up(popsize, n_threads);
    cuda_error_check(hipMalloc((void **)&d_random_states[1], n_blocks * n_threads * sizeof(hiprandState)));
    initialise_curand_kernel<<<n_blocks, n_threads>>>(d_random_states[1], 1);
    cuda_error_check(hipPeekAtLastError());
    cuda_error_check(hipDeviceSynchronize());

    int shared_bytes = next_power_of_two(dim) * 2 * sizeof(float);
    int block_size = next_power_of_two(dim);

    cuda_error_check(hipMalloc((void **)&indices[0], popsize * req_indices * sizeof(int)));
    cuda_error_check(hipMalloc((void **)&indices[1], popsize * req_indices * sizeof(int)));

    solution h_solution, *d_solution;
    cuda_error_check(hipMalloc((void **)&d_solution, sizeof(solution)));

    cuBenchmarkData cuBD(function_number, dim);
    std::vector<float> best_log;
    std::vector<float> time_log;
    std::vector<int> eval_log;

    // setup cuda timer to time the algorithm
    float time_ms;
    hipEvent_t start, stop;
    cuda_error_check(hipEventCreate(&start));
    cuda_error_check(hipEventCreate(&stop));

    for (int runs = 1; runs <= total_runs; runs++)
    {
        // reset solution
        cuda_error_check(hipMemset(d_solution, 0, sizeof(solution)));
        int i = 0, j = 1;
        shared_bytes = next_power_of_two(dim) * 2 * sizeof(float);

        // start timer 
        cuda_error_check(hipEventRecord(start, 0));
        
        population_initialisation_and_function_evaluation<<<blocks, block_size, shared_bytes, stream[i]>>>(
            function_number, x.get_device_ptr(), fitness.get_device_ptr(),
            dim, popsize, lower_bound, upper_bound, cuBD.get_shift_ptr(),
            cuBD.get_rotate_transpose_ptr(), cuBD.get_shuffle_ptr(), d_random_states[0],
            d_solution);

        select_random_trial_vectors<<<n_blocks, n_threads, 0, stream[j]>>>(indices[j], popsize, d_random_states[1]);

        cuda_error_check(hipPeekAtLastError());
        cuda_error_check(hipDeviceSynchronize());

        // // grab solution struct and see whether we stop
        cuda_error_check(hipMemcpy(&h_solution, d_solution, sizeof(solution), hipMemcpyDeviceToHost));

        int evals = popsize; // already evaluated parent population

        shared_bytes += next_power_of_two(dim) * sizeof(float) + sizeof(float);
        while (evals < max_evaluations && h_solution.solution_found == 0)
        {
            trial_vector_generation_function_evaluation_and_replacement<<<blocks, block_size, shared_bytes, stream[j]>>>(
                function_number, x.get_device_ptr(), fitness.get_device_ptr(),
                offspring.get_device_ptr(), offspring_fitness.get_device_ptr(),
                dim, popsize, indices[j], CR, F, lower_bound, upper_bound,
                cuBD.get_shift_ptr(), cuBD.get_rotate_transpose_ptr(),
                cuBD.get_shuffle_ptr(), d_random_states[0], d_solution);

            select_random_trial_vectors<<<n_blocks, n_threads, 0, stream[i]>>>(indices[i], popsize, d_random_states[1]);

            // grab solution struct and see whether we stop
            cuda_error_check(hipMemcpyAsync(&h_solution, d_solution, sizeof(solution), hipMemcpyDeviceToHost, stream[2]));

            cuda_error_check(hipPeekAtLastError());
            cuda_error_check(hipDeviceSynchronize());

            evals += popsize;
            swap(i, j);
        }
        // stop the timer
        cuda_error_check(hipEventRecord(stop, 0));
        cuda_error_check(hipEventSynchronize(stop));
        cuda_error_check(hipEventElapsedTime(&time_ms, start, stop));
        auto time_sec = time_ms / 1000; 

        fitness.cpu();
        
        auto best = *std::min_element(fitness.begin(), fitness.end());

        best_log.push_back(best);

        std::cout << "Function: " << function_number << " Best fitness: " << best << " Time: " << time_sec << std::endl;
        time_log.push_back(time_sec);
        eval_log.push_back(evals);
    }
    auto min_fitness = *std::min_element(best_log.begin(), best_log.end());
    auto max_fitness = *std::max_element(best_log.begin(), best_log.end());
    
    log_data(function_number, dim, popsize, "all_gpu_ide", best_log, time_log, eval_log); 

    float mean, stdev;
    mean_and_stdev(best_log, mean, stdev);
    float time_mean, time_stdev;
    mean_and_stdev(time_log, time_mean, time_stdev);
    float gen_mean, gen_stdev;
    mean_and_stdev(eval_log, gen_mean, gen_stdev);

    std::cout << function_number << ", " << min_fitness << ", " << max_fitness << ", " << mean
              << ", " << stdev << ", " << time_mean << ", " << time_stdev << ", "
              << gen_mean << ", " << gen_stdev << std::endl;

    hipFree(d_random_states[0]);
    hipFree(d_random_states[1]);
    hipFree(indices[0]);
    hipFree(indices[1]);
    for (int i = 0; i < 3; i++)
        hipStreamDestroy(stream[i]);
}